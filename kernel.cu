#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<vector>
#include<algorithm>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <random>
#include <cmath>
#include <iomanip>
#include "modules/utils.h"
#include <map>


using namespace std;

class hidden_layer 
{
public:
	map<char, vector<vector<double>>> gradient;
	map<char, vector<vector<double>>> params;
	int output_DIM;
	int input_DIM;
	int no_of_input;

	hidden_layer(int no_of_input, int input_DIM, int output_DIM)
	{
		this->input_DIM = input_DIM;
		this->no_of_input = no_of_input;
		this->output_DIM = output_DIM;
		this->params['W']= make2Dvec(input_DIM, output_DIM);
		this->params['b'] = make2Dvec(1, output_DIM);
		this->gradient['W']=make2Dvec(input_DIM, output_DIM);
		this->gradient['b'] = make2Dvec(1, output_DIM);
		weightInitialization();
	}

	vector<vector<double>> forward(vector<vector<double>> X)
	{
		auto forward_output = MATMUL(X, this->params['W']);
		vectoradd(forward_output, this->params['b']);
		return forward_output;
	}

	vector<vector<double>> backward(vector<vector<double>> X, vector<vector<double>> grad)
	{
		auto XT = MatTranspose(X);
		this->gradient['W'] = MATMUL(XT, grad);
		this->gradient['b'] = rowsum(grad);
		auto WT=MatTranspose(this->params['W']);
		auto backward_output = MATMUL(grad,WT);
		return backward_output;
	}
	void weightInitialization()
	{
		int  row = this->input_DIM;
		int col = this->output_DIM;
		float shape = row * col;
		random_device rd{};
		mt19937 gen{ rd() };
		normal_distribution<> d{ 0,1 };
		for (int i = 0; i < row; i++)
		{
			for (int j = 0; j < col; j++)
				this->params['W'][i][j] = double(d(gen)) * sqrtf(2/shape);
		}
	}
};


void miniBatchStochasticGradientDescent()
{
	;
}
int main()
{
	// model parameters
	int num_epoch = 10;
	int minibatch_size = 128;

	//optimisation alpha for momentum, lambda for weight decay
	float learning_rate = 0.01;
	int step = 10;
	float alpha = 0.99;
	float lambda = 0.01;
	float dropout = 0.5;
	string activation = "relu";
	
	if (activation == "relu")
		reLU act;
	else
		tanH act;

	// create objects for each layer
	int no_of_inputs = 3;
	int input_dimension = 2;
	int layer1_neurons = 3;
	int output_dimension = 1;
	hidden_layer L1(no_of_inputs, input_dimension, layer1_neurons);
	hidden_layer L2(no_of_inputs, layer1_neurons, output_dimension);

	string filename = "C:\\Users\\ragha\\source\\repos\\Neural Networks\\train-images.idx3-ubyte";

	//ReadMNIST(filename, 60000, 784, ar);

;

	return 0;
}

